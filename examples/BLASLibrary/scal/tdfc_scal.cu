// tdfc-cuda backend autocompiled body file
// tdfc version 1.160
// Thu May 26 17:01:38 2011


#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void tdfc_scal(float cc_alpha,float* cc_x,float* cc_x_out,int N  )
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx<N)
  {
          {
            cc_x_out[idx] = ((cc_x[idx]*cc_alpha));
          }
  }
} //tdfc_scal
