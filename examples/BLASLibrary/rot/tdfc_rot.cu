// tdfc-cuda backend autocompiled body file
// tdfc version 1.160
// Thu May 26 16:38:16 2011


#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void tdfc_rot(float cc_c,float cc_s,float* cc_x,float* cc_y,float* cc_x_out,float* cc_y_out,int N  )
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx<N)
  {
          {
            cc_x_out[idx] = (((cc_x[idx]*cc_c)+(cc_y[idx]*cc_s)));
            cc_y_out[idx] = (((cc_y[idx]*cc_c)-(cc_x[idx]*cc_s)));
          }
  }
} //tdfc_rot
