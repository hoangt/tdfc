// tdfc-cuda backend autocompiled body file
// tdfc version 1.160
// Thu May 26 15:56:56 2011


#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void tdfc_copy(float* cc_x,float* cc_y,int N  )
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx<N)
  {
          {
            cc_y[idx] = (cc_x[idx]);
          }
  }
} //tdfc_copy
