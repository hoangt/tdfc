// tdfc-cuda backend autocompiled body file
// tdfc version 1.160
// Fri May 27 17:47:08 2011


#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void tdfc_vadd(double cc_a,double* cc_x,double* cc_y,double* cc_z,int N  )
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx<N)
  {
          {
            cc_z[idx] = (((cc_a*cc_x[idx])+cc_y[idx]));
          }
  }
} //tdfc_vadd
