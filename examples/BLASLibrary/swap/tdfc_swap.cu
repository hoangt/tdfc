// tdfc-cuda backend autocompiled body file
// tdfc version 1.160
// Thu May 26 17:25:03 2011


#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void tdfc_swap(float* cc_x,float* cc_y,float* cc_x_out,float* cc_y_out,int N  )
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx<N)
  {
          {
            float cc_temp=cc_x[idx];
            cc_x_out[idx] = (cc_y[idx]);
            cc_y_out[idx] = (cc_temp);
          }
  }
} //tdfc_swap
