// tdfc-cuda backend autocompiled body file
// tdfc version 1.160
// Wed May 25 15:55:04 2011


#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void tdfc_axpy(float cc_alpha,float* cc_x,float* cc_y_in,float* cc_y_out,int N  )
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx<N)
  {
          {
            cc_y_out[idx] = (((cc_x[idx]*cc_alpha)+cc_y_in[idx]));
          }
  }
} //tdfc_axpy
